
#include <hip/hip_runtime.h>
/*
github: https://github.com/Daydream0929/cuda_index
blog: https://daydream0929.github.io/深入浅出cuda索引.html
*/

__global__ void test()
{
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    const int bid = by * blockDim.x + bx;
    const int block_tid = tz * (blockDim.x * blockDim.y) + ty * blockDim.x + tx;
    const int global_tid = bid * (blockDim.x * blockDim.y * blockDim.z) + block_tid;
    printf("bid: %d  --- block_tid : %d  --- globa_tid : %d\n", bid, block_tid, global_tid);
}

int main()
{
    dim3 blocks_per_grid = {2, 2, 1};
    dim3 threads_per_block = {2, 3, 4};
    test<<<blocks_per_grid, threads_per_block>>>();
    hipDeviceSynchronize();
    return 0;
}
